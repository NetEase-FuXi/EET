#include "hip/hip_runtime.h"
#include "core/common.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

// bert softmax code modified from Nvidia's DeepLearningExamples
// https://github.com/NVIDIA/DeepLearningExamples/blob/master/FasterTransformer/v3.1/fastertransformer/cuda/open_attention.cu#L1399-L1583

template <typename T>
__global__ void softmax_kernel_bert(T *qk_buf, const int64_t *padding_len, const int head_num, const int seq_len)
{
  int batch_id = blockIdx.x / head_num;
  int qk_offset = blockIdx.x * seq_len * seq_len;
  int mask_offset = batch_id * seq_len * seq_len;
  __shared__ float s_sum, s_max;

  int right_padding_len = 0;
  if (padding_len != nullptr)
  {
    right_padding_len = padding_len[batch_id];
  }

  for (int i = 0; i < seq_len - right_padding_len; ++i)
  {
    float qk = threadIdx.x < seq_len ? (float)qk_buf[threadIdx.x + qk_offset] : 0.0f;
    float padding_val = (threadIdx.x >= seq_len - right_padding_len) ? -1e20f : 0.0f;

    float tmp = threadIdx.x < seq_len ? (float)(qk + padding_val) : -1e20f;

    float max_val = blockReduceMax<float>(tmp);

    if (threadIdx.x == 0)
      s_max = max_val;
    __syncthreads();

    qk = threadIdx.x < seq_len ? __expf(tmp - s_max) : 0.0f;

    float sum_val = blockReduceSum<float>(qk);

    if (threadIdx.x == 0)
    {
      s_sum = sum_val + 1e-6f;
    }
    __syncthreads();

    if (threadIdx.x < seq_len)
      qk_buf[threadIdx.x + qk_offset] = (T)(qk / s_sum);

    qk_offset += seq_len;
  }
  for (int i = seq_len - right_padding_len; i < seq_len; ++i)
  {
    if (threadIdx.x < seq_len)
      qk_buf[threadIdx.x + qk_offset] = 0.0f;
    qk_offset += seq_len;
  }
}

template <typename T>
__global__ void softmax_kernel_bert_opt(T *qk_buf, const int64_t *padding_len, const int head_num, const int seq_len)
{
  int batch_id = blockIdx.x / (seq_len * head_num);
  int qk_offset = blockIdx.x * seq_len;
  
  int right_padding_len = 0;
  if (padding_len != nullptr)
  {
    right_padding_len = padding_len[batch_id];
  }

  __shared__ float s_sum, s_max;

  float qk = threadIdx.x < seq_len ? (float)qk_buf[threadIdx.x + qk_offset] : 0.0f;

  float tmp = threadIdx.x < seq_len - right_padding_len ? (float)(qk) : -1e20f;
  float max_val = blockReduceMax<float>(tmp);
  if (threadIdx.x == 0)
    s_max = max_val;
  __syncthreads();

  float qk_tmp = threadIdx.x < seq_len ? __expf((float)(tmp - s_max)) : 0.0f;
  float sum_val = blockReduceSum<float>(qk_tmp);

  if (threadIdx.x == 0)
  {
    s_sum = sum_val + 1e-6f;
  }
  __syncthreads();

  if (threadIdx.x < seq_len)
    qk_buf[threadIdx.x + qk_offset] = (T)(qk_tmp / s_sum);
}

template <typename T>
__global__ void softmax_kernel_v2(T *qk_buf, const int64_t *padding_len, const int head_num, const int seq_len)
{
  int batch_id = blockIdx.x / head_num;
  int qk_offset = blockIdx.x * seq_len * seq_len;
  int mask_offset = batch_id * seq_len * seq_len;
  __shared__ float s_sum, s_max;

  int right_padding_len = 0;
  if (padding_len != nullptr)
  {
    right_padding_len = padding_len[batch_id];
  }

  for (int i = 0; i < seq_len - right_padding_len; ++i)
  {
    float qk = threadIdx.x < seq_len ? (float)qk_buf[threadIdx.x + qk_offset] : 0.0f;
    float padding_val = (threadIdx.x > i || threadIdx.x >= seq_len - right_padding_len) ? -1e20f : 0.0f;

    float tmp = threadIdx.x < seq_len ? (float)(qk + padding_val) : -1e20f;
    float max_val = blockReduceMax<float>(tmp);

    if (threadIdx.x == 0)
      s_max = max_val;
    __syncthreads();

    qk = threadIdx.x < seq_len ? __expf(tmp - s_max) : 0.0f;

    float sum_val = blockReduceSum<float>(qk);

    if (threadIdx.x == 0)
    {
      s_sum = sum_val + 1e-6f;
    }
    __syncthreads();

    if (threadIdx.x < seq_len)
      qk_buf[threadIdx.x + qk_offset] = (T)(qk / s_sum);

    qk_offset += seq_len;
  }

  for (int i = seq_len - right_padding_len; i < seq_len; ++i)
  {
    if (threadIdx.x < seq_len)
      qk_buf[threadIdx.x + qk_offset] = 0.0f;
    qk_offset += seq_len;
  }
}

template <typename T>
__global__ void softmax_kernel_t5(T *qk_buf, T *position_bias, const int64_t *padding_len, const int head_num, const int seq_len)
{
  int batch_id = blockIdx.x / head_num;
  int head_id = blockIdx.x % head_num;
  int qk_offset = blockIdx.x * seq_len * seq_len;
  int mask_offset = batch_id * seq_len * seq_len;
  int bias_offset = head_id * seq_len * seq_len;
  __shared__ float s_sum, s_max;

  int right_padding_len = 0;
  if (padding_len != nullptr)
  {
    right_padding_len = padding_len[batch_id];
  }

  for (int i = 0; i < seq_len - right_padding_len; ++i)
  {
    float qk = threadIdx.x < seq_len ? (float)qk_buf[threadIdx.x + qk_offset] + (float)position_bias[threadIdx.x + bias_offset] : 0.0f;
    float padding_val = (threadIdx.x >= seq_len - right_padding_len) ? -1e20f : 0.0f;

    float tmp = threadIdx.x < seq_len ? (float)(qk + padding_val) : -1e20f;

    float max_val = blockReduceMax<float>(tmp);

    if (threadIdx.x == 0)
      s_max = max_val;
    __syncthreads();

    qk = threadIdx.x < seq_len ? __expf(tmp - s_max) : 0.0f;

    float sum_val = blockReduceSum<float>(qk);

    if (threadIdx.x == 0)
    {
      s_sum = sum_val + 1e-6f;
    }
    __syncthreads();

    if (threadIdx.x < seq_len)
      qk_buf[threadIdx.x + qk_offset] = (T)(qk / s_sum);

    qk_offset += seq_len;
    bias_offset += seq_len;
  }
  for (int i = seq_len - right_padding_len; i < seq_len; ++i)
  {
    if (threadIdx.x < seq_len)
      qk_buf[threadIdx.x + qk_offset] = 0.0f;
    qk_offset += seq_len;
    bias_offset += seq_len;
  }
}

template <typename T>
__global__ void softmax_kernel_t5_opt(T *qk_buf, T *position_bias, const int64_t *padding_len, const int head_num, const int seq_len)
{
  int batch_id = blockIdx.x / (seq_len * head_num);
  int qk_offset = blockIdx.x * seq_len;
  int bias_offset = blockIdx.x % (seq_len * head_num) * seq_len;
  
  int right_padding_len = 0;
  if (padding_len != nullptr)
  {
    right_padding_len = padding_len[batch_id];
  }

  __shared__ float s_sum, s_max;

  float qk = threadIdx.x < seq_len ? (float)qk_buf[threadIdx.x + qk_offset] + (float)position_bias[threadIdx.x + bias_offset] : 0.0f;

  float tmp = threadIdx.x < seq_len - right_padding_len ? (float)(qk) : -1e20f;
  float max_val = blockReduceMax<float>(tmp);
  if (threadIdx.x == 0)
    s_max = max_val;
  __syncthreads();

  float qk_tmp = threadIdx.x < seq_len ? __expf((float)(tmp - s_max)) : 0.0f;
  float sum_val = blockReduceSum<float>(qk_tmp);

  if (threadIdx.x == 0)
  {
    s_sum = sum_val + 1e-6f;
  }
  __syncthreads();

  if (threadIdx.x < seq_len)
    qk_buf[threadIdx.x + qk_offset] = (T)(qk_tmp / s_sum);
}

template <class T>
void bert_softmax_kernel(void *qk_buf, void* position_bias, const int64_t *padding_len, const int &batch_size, const int &head_num,
                         const int &seq_len, bool need_sequence_mask, const hipStream_t stream)
{
  dim3 grid, block;

  if (seq_len <= 32)
    block.x = 32;
  else if (seq_len > 32 && seq_len <= 64)
    block.x = 64;
  else if (seq_len > 64 && seq_len <= 128)
    block.x = 128;
  else if (seq_len > 128 && seq_len <= 256)
    block.x = 256;
  else if (seq_len > 256 && seq_len <= 512)
    block.x = 512;
  else
    block.x = 1024;

  if (need_sequence_mask) {
    grid.x = batch_size * head_num;
    softmax_kernel_v2<T><<<grid, block, 0, stream>>>((T*)qk_buf, padding_len, head_num, seq_len);
  } else {
    if (position_bias == nullptr) {
      if (seq_len >= 32) {
        grid.x = batch_size * head_num * seq_len;
        softmax_kernel_bert_opt<T><<<grid, block, 0, stream>>>((T *)qk_buf, padding_len, head_num, seq_len);
      } else {
        grid.x = batch_size * head_num;
        softmax_kernel_bert<T><<<grid, block, 0, stream>>>((T *)qk_buf,padding_len, head_num, seq_len);
      }
    } else {
      if (seq_len >= 32) {
        grid.x = batch_size * head_num * seq_len;
        softmax_kernel_t5_opt<T><<<grid, block, 0, stream>>>((T *)qk_buf, (T*)position_bias, padding_len, head_num, seq_len);
      } else {
        grid.x = batch_size * head_num;
        softmax_kernel_t5<T><<<grid, block, 0, stream>>>((T *)qk_buf, (T*)position_bias,padding_len, head_num, seq_len);
      }      
    }
  }
}

template void bert_softmax_kernel<float>(void *qk_buf, void* position_bias, const int64_t *padding_len, const int &batch_size,
                                         const int &head_num, const int &seq_len, bool need_sequence_mask, const hipStream_t stream);
template void bert_softmax_kernel<half>(void *qk_buf, void* position_bias, const int64_t *padding_len, const int &batch_size,
                                        const int &head_num, const int &seq_len, bool need_sequence_mask, const hipStream_t stream);
