#include "hip/hip_runtime.h"
#include "core/common.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

// bert softmax code modified from Nvidia's DeepLearningExamples
// https://github.com/NVIDIA/DeepLearningExamples/blob/master/FasterTransformer/v3.1/fastertransformer/cuda/open_attention.cu#L1399-L1583

template <typename T>
__global__ void softmax_kernel_bert(T *qk_buf, const int64_t *padding_len, const int head_num, const int seq_len, const T scalar)
{
  int batch_id = blockIdx.x / head_num;
  int qk_offset = blockIdx.x * seq_len * seq_len;
  int mask_offset = batch_id * seq_len * seq_len;
  __shared__ float s_sum, s_max;

  int right_padding_len = 0;
  if (padding_len != nullptr)
  {
    right_padding_len = padding_len[batch_id];
  }

  for (int i = 0; i < seq_len - right_padding_len; ++i)
  {
    float qk = threadIdx.x < seq_len ? (float)qk_buf[threadIdx.x + qk_offset] : 0.0f;
    float padding_val = (threadIdx.x >= seq_len - right_padding_len) ? -1e20f : 0.0f;

    float tmp = threadIdx.x < seq_len ? (float)(qk * (float)scalar + padding_val) : -1e20f;

    float max_val = blockReduceMax<float>(tmp);

    if (threadIdx.x == 0)
      s_max = max_val;
    __syncthreads();

    qk = threadIdx.x < seq_len ? __expf(tmp - s_max) : 0.0f;

    float sum_val = blockReduceSum<float>(qk);

    if (threadIdx.x == 0)
    {
      s_sum = sum_val + 1e-6f;
    }
    __syncthreads();

    if (threadIdx.x < seq_len)
      qk_buf[threadIdx.x + qk_offset] = (T)(qk / s_sum);

    qk_offset += seq_len;
  }
  for (int i = seq_len - right_padding_len; i < seq_len; ++i)
  {
    if (threadIdx.x < seq_len)
      qk_buf[threadIdx.x + qk_offset] = 0.0f;
    qk_offset += seq_len;
  }
}

template <typename T>
__global__ void softmax_kernel_v2(T *qk_buf, const int64_t *padding_len, const int head_num, const int seq_len, const T scalar)
{
  int batch_id = blockIdx.x / head_num;
  int qk_offset = blockIdx.x * seq_len * seq_len;
  int mask_offset = batch_id * seq_len * seq_len;
  __shared__ float s_sum, s_max;

  int right_padding_len = 0;
  if (padding_len != nullptr)
  {
    right_padding_len = padding_len[batch_id];
  }

  for (int i = 0; i < seq_len - right_padding_len; ++i)
  {
    float qk = threadIdx.x < seq_len ? (float)qk_buf[threadIdx.x + qk_offset] : 0.0f;
    float padding_val = (threadIdx.x > i || threadIdx.x >= seq_len - right_padding_len) ? -1e20f : 0.0f;

    float tmp = threadIdx.x < seq_len ? (float)(qk * (float)scalar + padding_val) : -1e20f;
    float max_val = blockReduceMax<float>(tmp);

    if (threadIdx.x == 0)
      s_max = max_val;
    __syncthreads();

    qk = threadIdx.x < seq_len ? __expf(tmp - s_max) : 0.0f;

    float sum_val = blockReduceSum<float>(qk);

    if (threadIdx.x == 0)
    {
      s_sum = sum_val + 1e-6f;
    }
    __syncthreads();

    if (threadIdx.x < seq_len)
      qk_buf[threadIdx.x + qk_offset] = (T)(qk / s_sum);

    qk_offset += seq_len;
  }

  for (int i = seq_len - right_padding_len; i < seq_len; ++i)
  {
    if (threadIdx.x < seq_len)
      qk_buf[threadIdx.x + qk_offset] = 0.0f;
    qk_offset += seq_len;
  }
}

template <class T>
void bert_softmax_kernel(void *qk_buf, const int64_t *padding_len, const int &batch_size, const int &head_num,
                         const int &seq_len, const float &scalar, bool need_sequence_mask, const hipStream_t stream)
{
  dim3 grid, block;

  if (seq_len <= 32)
    block.x = 32;
  else if (seq_len > 32 && seq_len <= 64)
    block.x = 64;
  else if (seq_len > 64 && seq_len <= 128)
    block.x = 128;
  else if (seq_len > 128 && seq_len <= 256)
    block.x = 256;
  else if (seq_len > 256 && seq_len <= 512)
    block.x = 512;
  else
    block.x = 1024;

  grid.x = batch_size * head_num;
  if (need_sequence_mask) {
    softmax_kernel_v2<T><<<grid, block, 0, stream>>>((T*)qk_buf, padding_len, head_num, seq_len, scalar);
  } else {
    softmax_kernel_bert<T><<<grid, block, 0, stream>>>((T *)qk_buf, padding_len, head_num, seq_len, scalar);
  }
}

template void bert_softmax_kernel<float>(void *qk_buf, const int64_t *padding_len, const int &batch_size,
                                         const int &head_num, const int &seq_len, const float &scalar, bool need_sequence_mask, const hipStream_t stream);
template void bert_softmax_kernel<half>(void *qk_buf, const int64_t *padding_len, const int &batch_size,
                                        const int &head_num, const int &seq_len, const float &scalar, bool need_sequence_mask, const hipStream_t stream);
