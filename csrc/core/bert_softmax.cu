#include "hip/hip_runtime.h"
#include "core/common.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

constexpr int SOFTMAX_BLOCK_SIZE = 128;
constexpr int WARP_SIZE = 32;

template <typename T>
__global__ void bert_softmax_kernel(T *qk_buf, const int64_t *padding_len, const int head_num, const int seq_len)
{
  int batch_id = blockIdx.x / (seq_len * head_num);
  int qk_offset = blockIdx.x * seq_len;
  __shared__ float s_sum, s_max;
  
  int right_padding_len = 0;
  if (padding_len != nullptr)
  {
    right_padding_len = padding_len[batch_id];
  }

  float qk = threadIdx.x < seq_len - right_padding_len ? static_cast<float>(qk_buf[threadIdx.x + qk_offset]) : -1e20f;
  float max_val = blockReduceMax<float>(qk);
  if (threadIdx.x == 0)
    s_max = max_val;
  __syncthreads();

  float qk_tmp = threadIdx.x < seq_len - right_padding_len ? __expf((float)(qk - s_max)) : 0.0f;
  float sum_val = blockReduceSum<float>(qk_tmp);

  if (threadIdx.x == 0)
  {
    s_sum = sum_val + 1e-6f;
  }
  __syncthreads();

  if (threadIdx.x < seq_len)
    qk_buf[threadIdx.x + qk_offset] = (T)(qk_tmp / s_sum);
}

template <typename T, int block_size>
__global__ void bert_softmax_kernel_v2(T *qk_buf, const int64_t *padding_len, const int head_num, const int seq_len)
{
  const int tid = threadIdx.x;
  int batch_id = blockIdx.x / (seq_len * head_num);
  int qk_offset = blockIdx.x * seq_len;

  extern __shared__ __align__(sizeof(double)) unsigned char shared_buf[];
  auto* buf = reinterpret_cast<float*>(shared_buf);
  __shared__ float s_sum, s_max;

  int right_padding_len = 0;
  if (padding_len != nullptr)
  {
    right_padding_len = padding_len[batch_id];
  }

  // load & max
  float thread_max = -1e20f;
  for (int col_id = tid; col_id < seq_len - right_padding_len; col_id += block_size) {
    buf[col_id] = static_cast<float>(qk_buf[qk_offset + col_id]);
    thread_max = max(thread_max, buf[col_id]);
  }
  float max_val = blockReduceMax<float>(thread_max);
  if (threadIdx.x == 0)
    s_max = max_val;
  __syncthreads();

  // sum
  float thread_sum = 0;
  for (int col_id = tid; col_id < seq_len - right_padding_len; col_id += block_size) {
    float exp_x = __expf(buf[col_id] - s_max);
    buf[col_id] = exp_x;
    thread_sum += exp_x;
  }
  float sum_val = blockReduceSum<float>(thread_sum);

  if (threadIdx.x == 0)
  {
    s_sum = sum_val + 1e-6f;
  }
  __syncthreads();

  // store
  for (int col_id = tid; col_id < seq_len; col_id += block_size) {
    qk_buf[qk_offset + col_id] = col_id < seq_len - right_padding_len ? (T)(buf[col_id] / s_sum) : (T)0.0f;
  }
}

template <typename T>
__global__ void masked_softmax_kernel(T *qk_buf, const int64_t *padding_len, const int head_num, const int seq_len)
{
  int batch_id = blockIdx.x / head_num;
  int qk_offset = blockIdx.x * seq_len * seq_len;
  int mask_offset = batch_id * seq_len * seq_len;
  __shared__ float s_sum, s_max;

  int right_padding_len = 0;
  if (padding_len != nullptr)
  {
    right_padding_len = padding_len[batch_id];
  }

  for (int i = 0; i < seq_len - right_padding_len; ++i)
  {
    float qk = threadIdx.x < seq_len ? static_cast<float>(qk_buf[threadIdx.x + qk_offset]) : 0.0f;
    float padding_val = (threadIdx.x > i || threadIdx.x >= seq_len - right_padding_len) ? -1e20f : 0.0f;

    float tmp = threadIdx.x < seq_len ? (float)(qk + padding_val) : -1e20f;
    float max_val = blockReduceMax<float>(tmp);

    if (threadIdx.x == 0)
      s_max = max_val;
    __syncthreads();

    qk = threadIdx.x < seq_len ? __expf(tmp - s_max) : 0.0f;

    float sum_val = blockReduceSum<float>(qk);

    if (threadIdx.x == 0)
    {
      s_sum = sum_val + 1e-6f;
    }
    __syncthreads();

    if (threadIdx.x < seq_len)
      qk_buf[threadIdx.x + qk_offset] = (T)(qk / s_sum);

    qk_offset += seq_len;
  }

  for (int i = seq_len - right_padding_len; i < seq_len; ++i)
  {
    if (threadIdx.x < seq_len)
      qk_buf[threadIdx.x + qk_offset] = (T)0.0f;
    qk_offset += seq_len;
  }
}

template <typename T>
__global__ void t5_softmax_kernel(T *qk_buf, T *position_bias, const int64_t *padding_len, const int head_num, const int seq_len)
{
  int batch_id = blockIdx.x / (seq_len * head_num);
  int qk_offset = blockIdx.x * seq_len;
  int bias_offset = blockIdx.x % (seq_len * head_num) * seq_len;
  __shared__ float s_sum, s_max;

  int right_padding_len = 0;
  if (padding_len != nullptr)
  {
    right_padding_len = padding_len[batch_id];
  }

  float qk = threadIdx.x < seq_len - right_padding_len ? static_cast<float>(qk_buf[threadIdx.x + qk_offset]) + static_cast<float>(position_bias[threadIdx.x + bias_offset]) : -1e20f;
  float max_val = blockReduceMax<float>(qk);
  if (threadIdx.x == 0)
    s_max = max_val;
  __syncthreads();

  float qk_tmp = threadIdx.x < seq_len - right_padding_len ? __expf((float)(qk - s_max)) : 0.0f;
  float sum_val = blockReduceSum<float>(qk_tmp);

  if (threadIdx.x == 0)
  {
    s_sum = sum_val + 1e-6f;
  }
  __syncthreads();

  if (threadIdx.x < seq_len)
    qk_buf[threadIdx.x + qk_offset] = (T)(qk_tmp / s_sum);
}

template <typename T, int block_size>
__global__ void t5_softmax_kernel_v2(T *qk_buf, T *position_bias, const int64_t *padding_len, const int head_num, const int seq_len)
{
  const int tid = threadIdx.x;
  int batch_id = blockIdx.x / (seq_len * head_num);
  int qk_offset = blockIdx.x * seq_len;
  int bias_offset = blockIdx.x % (seq_len * head_num) * seq_len;

  extern __shared__ __align__(sizeof(double)) unsigned char shared_buf[];
  auto* buf = reinterpret_cast<float*>(shared_buf);
  __shared__ float s_sum, s_max;

  int right_padding_len = 0;
  if (padding_len != nullptr)
  {
    right_padding_len = padding_len[batch_id];
  }

  // load & max
  float thread_max = -1e20f;
  for (int col_id = tid; col_id < seq_len - right_padding_len; col_id += block_size) {
    buf[col_id] = static_cast<float>(qk_buf[qk_offset + col_id]) + static_cast<float>(position_bias[bias_offset + col_id]);
    thread_max = max(thread_max, buf[col_id]);
  }
  float max_val = blockReduceMax<float>(thread_max);
  if (threadIdx.x == 0)
    s_max = max_val;
  __syncthreads();

  // sum
  float thread_sum = 0;
  for (int col_id = tid; col_id < seq_len - right_padding_len; col_id += block_size) {
    float exp_x = __expf(buf[col_id] - s_max);
    buf[col_id] = exp_x;
    thread_sum += exp_x;
  }
  float sum_val = blockReduceSum<float>(thread_sum);

  if (threadIdx.x == 0)
  {
    s_sum = sum_val + 1e-6f;
  }
  __syncthreads();

  // store
  for (int col_id = tid; col_id < seq_len; col_id += block_size) {
    qk_buf[qk_offset + col_id] = col_id < seq_len - right_padding_len ? (T)(buf[col_id] / s_sum) : (T)0.0f;
  }
}

template <class T>
void launch_softmax_kernel(void *qk_buf, void* position_bias, const int64_t *padding_len, const int batch_size, const int head_num,
                              const int seq_len, bool need_sequence_mask, const hipStream_t stream)
{
  const size_t smem_size = seq_len * sizeof(float);
  const int grid_dim_x = batch_size * head_num * seq_len;
  int block_dim_x;

  assert(seq_len <= 2048);
  if (need_sequence_mask) {
    block_dim_x = min(((seq_len + 31) / 32) * 32, 1024);
    masked_softmax_kernel<T><<<grid_dim_x, block_dim_x, 0, stream>>>((T*)qk_buf, padding_len, head_num, seq_len);
  } else {
    if (position_bias == nullptr) {
      if (seq_len <= 128) {
        block_dim_x = min(((seq_len + 31) / 32) * 32, 1024);
        bert_softmax_kernel<T><<<grid_dim_x, block_dim_x, 0, stream>>>((T*)qk_buf, padding_len, head_num, seq_len);
      } else {
        block_dim_x = SOFTMAX_BLOCK_SIZE;
        bert_softmax_kernel_v2<T, SOFTMAX_BLOCK_SIZE><<<grid_dim_x, block_dim_x, smem_size, stream>>>((T *)qk_buf, padding_len, head_num, seq_len);
      }
    } else {
      if (seq_len <= 128) {
        block_dim_x = min(((seq_len + 31) / 32) * 32, 1024);
        t5_softmax_kernel<T><<<grid_dim_x, block_dim_x, 0, stream>>>((T *)qk_buf, (T*)position_bias, padding_len, head_num, seq_len);
      } else {
        block_dim_x = SOFTMAX_BLOCK_SIZE;
        t5_softmax_kernel_v2<T, SOFTMAX_BLOCK_SIZE><<<grid_dim_x, block_dim_x, smem_size, stream>>>((T *)qk_buf, (T*)position_bias, padding_len, head_num, seq_len);
      }
    }
  }
}

template void launch_softmax_kernel<float>(void *qk_buf, void *position_bias, const int64_t *padding_len, const int batch_size,
                                           const int head_num, const int seq_len, bool need_sequence_mask, const hipStream_t stream);
template void launch_softmax_kernel<half>(void *qk_buf, void *position_bias, const int64_t *padding_len, const int batch_size,
                                          const int head_num, const int seq_len, bool need_sequence_mask, const hipStream_t stream);
