#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <iostream>
// constants for approximating the normal cdf
// gelu ->gelu_fast
constexpr static float A = 0.5;
constexpr static float B = 0.7978845608028654;   // sqrt(2.0/M_PI)
constexpr static float C = 0.035677408136300125; // 0.044715 * sqrt(2.0/M_PI)
constexpr static float D = 1.702;

template <typename T>
__global__
void add_bias_gelu(T* out, const T* bias, int m, int n)
{
  int idx = n * blockIdx.x + blockIdx.y * blockDim.x + threadIdx.x;
  int bias_idx = blockIdx.y * blockDim.x + threadIdx.x;

  if (idx < m * n){
    T in = out[idx] + bias[bias_idx];
    T cdf = A + A * tanh(in * (C * in * in + B));
    out[idx] = in * cdf;
  }
}

template <>
__global__
void add_bias_gelu<half>(half* out, const half* bias, int m, int n)
{
  const half2 A2 = __floats2half2_rn(A, A);
  const half2 B2 = __floats2half2_rn(B, B);
  const half2 C2 = __floats2half2_rn(C, C);

  half2 * out_ptr = (half2 *)out;
  half2 * bias_ptr = (half2 *)bias;

  int idx = n * blockIdx.x + blockIdx.y * blockDim.x + threadIdx.x;
  int bias_idx = blockIdx.y * blockDim.x + threadIdx.x;

  if (idx < m * n ){
    half2 in = out_ptr[idx] + bias_ptr[bias_idx];
    half2 tmp = in * (C2 * in * in + B2);
    float x = tanh(__half2float(reinterpret_cast<half&>(tmp.x)));
    float y = tanh(__half2float(reinterpret_cast<half&>(tmp.y)));
    half2 cdf = A2 + A2 * make_half2(x, y);
    out_ptr[idx] = in * cdf;
  }
}

template <typename T>
__global__
void add_bias_quick_gelu(T* out, const T* bias, int m, int n) 
{
  int idx = n * blockIdx.x + blockIdx.y * blockDim.x + threadIdx.x;
  int bias_idx = blockIdx.y * blockDim.x + threadIdx.x;

  if (idx < m * n) {
    T in = out[idx] + bias[bias_idx];
    T cdf = 1.0f / (1.0f + __expf(-(D * in)));
    out[idx] = in * cdf;
  }
}

template <>
__global__
void add_bias_quick_gelu(half* out, const half* bias, int m, int n)
{
  const half2 D2 = __floats2half2_rn(D, D);
  const half2 half_one2 = __floats2half2_rn(1.0f, 1.0f);

  half2 *out_ptr = (half2 *)out;
  half2 *bias_ptr = (half2 *)bias;

  int idx = n * blockIdx.x + blockIdx.y * blockDim.x + threadIdx.x;
  int bias_idx = blockIdx.y * blockDim.x + threadIdx.x;

  if (idx < m * n) {
    half2 in = out_ptr[idx] + bias_ptr[bias_idx];
    half2 tmp = __hmul2(D2, in);
    float x = __expf(__half2float(-reinterpret_cast<half&>(tmp.x)));
    float y = __expf(__half2float(-reinterpret_cast<half&>(tmp.y)));
    half2 cdf = __h2div(half_one2, half_one2 + make_half2(x, y));
    out_ptr[idx] = __hmul2(in, cdf);
  }
}

template <typename T>
__global__
void add_bias_relu(T* out, const T* bias, int m, int n)
{
    int idx = n * blockIdx.x + blockIdx.y * blockDim.x + threadIdx.x;
    int bias_idx = blockIdx.y * blockDim.x + threadIdx.x;

    if (idx < m * n){
        T val = out[idx] + bias[bias_idx];
        out[idx] = (T)(val > 0.0f ? val : 0.0f);
    }
}

template <>
  __global__ 
void add_bias_relu(half* out, const half* bias, int m, int n)
{
    int idx =  n * blockIdx.x + blockIdx.y * blockDim.x + threadIdx.x;
    int bias_idx = blockIdx.y * blockDim.x + threadIdx.x;

    half2 *out_ptr = (half2 *)out;
    half2 *bias_ptr = (half2 *)bias;
    if (bias_idx < n && idx < m * n){
        half2 val = __hadd2(out_ptr[idx], bias_ptr[bias_idx]);
        reinterpret_cast<half&>(val.x) = reinterpret_cast<half&>(val.x) > (half)0.0f ? reinterpret_cast<half&>(val.x) : (half)0.0f;
        reinterpret_cast<half&>(val.y) = reinterpret_cast<half&>(val.y) > (half)0.0f ? reinterpret_cast<half&>(val.y) : (half)0.0f;
        out_ptr[idx] = val;
    }
}

template<typename T>
void add_bias_act_kernel(void* ffn_inner, const void* bias, int m, int n ,const int act_type ,const hipStream_t stream)
{
  if (sizeof(T) == sizeof(half)){

    int fold_coeff = 1;
    if (n <= 2048){
      fold_coeff = 1;
    }else if( n <= 4096){
      fold_coeff = 2;
    }else if(n <= 8192){
      fold_coeff = 4;
    }else if(n <= 16384){
      fold_coeff = 8;
    }else if(n <= 16384 * 2){
      fold_coeff = 16;
    }else if(n <= 16384 * 4){
      fold_coeff = 32;
    }
  
    dim3 grid(m, fold_coeff);
    dim3 block(n / fold_coeff);
  

    block.x /= 2;
    if (act_type == 0) {
      add_bias_relu<T><<<grid, block, 0, stream>>>((T *)ffn_inner, (T *)bias, m, n / 2);
    } else if (act_type == 1) {
      add_bias_gelu<T><<<grid, block, 0, stream>>>((T *)ffn_inner, (T *)bias, m, n / 2);
    } else if (act_type ==2) {
      add_bias_quick_gelu<T><<<grid, block, 0, stream>>>((T *)ffn_inner, (T *)bias, m, n / 2);
    }
    else {
      std::cerr << "unsupported activation " << std::endl;
    }
  } else {

    int fold_coeff = 1;
    if (n <= 1024){
      fold_coeff = 1;
    }else if( n <= 2048){
      fold_coeff = 2;
    }else if(n <= 4096){
      fold_coeff = 4;
    }else if(n <= 8192){
      fold_coeff = 8;
    }else if(n <= 16384){
      fold_coeff = 16;
    }else if(n <= 16384 * 2){
      fold_coeff = 32;
    }else if (n <= 16384 * 4){
      fold_coeff = 64;
    }
  
    dim3 grid(m, fold_coeff);
    dim3 block(n / fold_coeff);
  

    if (act_type == 0) {
      add_bias_relu<T><<<grid, block, 0, stream>>>((T *)ffn_inner, (T *)bias, m, n);
    } else if (act_type == 1) {
      add_bias_gelu<T><<<grid, block, 0, stream>>>((T *)ffn_inner, (T *)bias, m, n);
    } else if (act_type == 2) {
      add_bias_quick_gelu<T><<<grid, block, 0, stream>>>((T *)ffn_inner, (T *)bias, m, n);
    } else {
      std::cerr << "unsupported activation " << std::endl;
    }
  }
}

template void add_bias_act_kernel<float>(void* ffn_inner, const void* bias, const int m, const int n, const int act_type, const hipStream_t stream);
template void add_bias_act_kernel<half>(void* ffn_inner, const void* bias, const int m, const int n, const int act_type, const hipStream_t stream);
