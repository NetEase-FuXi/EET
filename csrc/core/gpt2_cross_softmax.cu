#include "hip/hip_runtime.h"
#include "core/common.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <vector>

// cross softmax code modified from Nvidia's DeepLearningExamples
// https://github.com/NVIDIA/DeepLearningExamples/blob/master/FasterTransformer/v3.1/fastertransformer/cuda/open_attention.cu#L1399-L1583


template <typename T>
__global__
void cross_softmax_kernel_opt(T *qk_buf_, const int head_num, const int seq_len,const int mem_seq_len, const T scalar)
{
    int qk_offset = blockIdx.x * seq_len * mem_seq_len;

    __shared__ float s_sum, s_max;

    for(int i = 0; i < seq_len; ++i)
    {
        float qk = threadIdx.x < mem_seq_len ? (float)qk_buf_[threadIdx.x + qk_offset] : 0.0f;

        float tmp = threadIdx.x < mem_seq_len ? (float)(qk * (float)scalar): -1e20f;

        float max_val = blockReduceMax<float>(tmp);

        if(threadIdx.x == 0)
            s_max = max_val;
        __syncthreads();

        qk = threadIdx.x < mem_seq_len ? __expf(tmp - s_max) : 0.0f;

        float sum_val = blockReduceSum<float>(qk);

        if(threadIdx.x == 0)
        {
            s_sum = sum_val + 1e-6f;
        }
        __syncthreads();

        if(threadIdx.x < mem_seq_len)
            qk_buf_[threadIdx.x + qk_offset] = (T)(qk / s_sum);

        qk_offset += mem_seq_len;
    }
}

template <class T>
void cross_softmax_kernel(void *qk_buf_, const int &batch_size,
                          const int &head_num, const int &seq_len, const int &mem_seq_len, const float &scalar, const hipStream_t stream)
{
  dim3 grid, block;

  if (mem_seq_len <= 32)
    block.x = 32;
  else if (mem_seq_len > 32 && mem_seq_len <= 64)
    block.x = 64;
  else if (mem_seq_len > 64 && mem_seq_len <= 128)
    block.x = 128;
  else if (mem_seq_len > 128 && mem_seq_len <= 256)
    block.x = 256;
  else if (mem_seq_len > 256 && mem_seq_len <= 512)
    block.x = 512;
  else
    block.x = 1024;

  grid.x = batch_size * head_num;
  cross_softmax_kernel_opt<T><<<grid, block, 0, stream>>>((T *)qk_buf_, head_num, seq_len, mem_seq_len, scalar);
}

template void cross_softmax_kernel<float>(void *qk_buf_,const int& batch_size, 
                                      const int& head_num, const int& seq_len,const int& mem_seq_len, const float& scalar, const hipStream_t stream);

template void cross_softmax_kernel<half>(void *qk_buf_, const int& batch_size, 
                                      const int& head_num, const int& seq_len,const int& mem_seq_len, const float& scalar, const hipStream_t stream);

