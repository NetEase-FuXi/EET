#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include "hipcub/hipcub.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <c10/cuda/CUDAGuard.h>
#include "fpA_intB_gemm_wrapper.h"
#include "fpA_intB_gemm.h"
#include "cutlass_preprocessors.h"
#include "cuda_utils.h"
#include "torch_utils.h"

#include <vector>

namespace ft = fastertransformer;

int getWorkspaceSize(const int m, const int n, const int k)
{
    // These are the min tile sizes for each config, which would launch the maximum number of blocks
    const int max_grid_m = (m + 31) / 32;
    const int max_grid_n = (n + 127) / 128;
    const int split_k_limit = 7;
    // We need 4 bytes per block in the worst case. We launch split_k_limit in z dim.
    return max_grid_m * max_grid_n * split_k_limit * 4;
}

std::vector<torch::Tensor>
symmetric_quantize_last_axis_of_tensor(torch::Tensor &weight,
                                       py::object &quant_type,
                                       bool return_unprocessed_quantized_tensor)
{
    CHECK_CPU(weight);
    CHECK_CONTIGUOUS(weight);
    TORCH_CHECK(weight.numel() != 0, "weight should not be empty tensor");
    TORCH_CHECK(weight.dim() == 2 || weight.dim() == 3, "Invalid dim. The dim of weight should be 2 or 3");

    torch::ScalarType _quant_type = torch::python::detail::py_object_to_dtype(quant_type);
    auto _st = weight.scalar_type();
    TORCH_CHECK(_st == torch::kFloat32 || _st == torch::kFloat16, "Invalid datatype. Weight must be FP16 or FP32");
    TORCH_CHECK(_quant_type == torch::kInt8 || _quant_type == at::ScalarType::QUInt4x2, "Must be int4 or int8 quantization");
    ft::QuantType ft_quant_type = ft::get_ft_quant_type(_quant_type);

    const size_t num_experts = weight.dim() == 2 ? 1 : weight.size(0);
    const size_t num_rows    = weight.size(-2);
    const size_t num_cols    = weight.size(-1);

    const size_t bits_in_type      = ft::get_bits_in_quant_type(ft_quant_type);
    const size_t bytes_per_out_col = num_cols * bits_in_type / 8;

    const size_t input_mat_size     = num_rows * num_cols;
    const size_t quantized_mat_size = num_rows * bytes_per_out_col;

    std::vector<long int> quantized_weight_shape;
    std::vector<long int> scale_shape;
    if (weight.dim() == 2) {
        quantized_weight_shape = {long(num_rows), long(bytes_per_out_col)};
        scale_shape            = {long(num_cols)};
    }
    else if (weight.dim() == 3) {
        quantized_weight_shape = {long(num_experts), long(num_rows), long(bytes_per_out_col)};
        scale_shape            = {long(num_experts), long(num_cols)};
    }
    else {
        TORCH_CHECK(false, "Invalid weight dimension. Weight must have dim 2 or 3");
    }

    torch::Tensor unprocessed_quantized_weight =
        torch::empty(quantized_weight_shape, torch::dtype(torch::kInt8).device(torch::kCPU).requires_grad(false));

    torch::Tensor processed_quantized_weight = torch::empty_like(unprocessed_quantized_weight);

    torch::Tensor scales = torch::empty(scale_shape, torch::dtype(weight.dtype()).device(torch::kCPU).requires_grad(false));

    int8_t *unprocessed_quantized_weight_ptr = reinterpret_cast<int8_t *>(unprocessed_quantized_weight.data_ptr());
    int8_t *processed_quantized_weight_ptr = reinterpret_cast<int8_t *>(processed_quantized_weight.data_ptr());

    if (weight.scalar_type() == at::ScalarType::Float)
    {
        ft::symmetric_quantize<float, float>(processed_quantized_weight_ptr,
                                             unprocessed_quantized_weight_ptr,
                                             reinterpret_cast<float *>(scales.data_ptr()),
                                             reinterpret_cast<const float *>(weight.data_ptr()),
                                             {num_rows, num_cols},
                                             ft_quant_type);
    }
    else if (weight.scalar_type() == at::ScalarType::Half)
    {
        ft::symmetric_quantize<half, half>(processed_quantized_weight_ptr,
                                           unprocessed_quantized_weight_ptr,
                                           reinterpret_cast<half *>(scales.data_ptr()),
                                           reinterpret_cast<const half *>(weight.data_ptr()),
                                           {num_rows, num_cols},
                                           ft_quant_type);
    }
    else
    {
        TORCH_CHECK(false, "Invalid data type. Weight must be FP32/FP16");
    }

    if (return_unprocessed_quantized_tensor)
    {
        return std::vector<torch::Tensor>{unprocessed_quantized_weight, processed_quantized_weight, scales};
    }

    return std::vector<torch::Tensor>{processed_quantized_weight, scales};
}

torch::Tensor preprocess_weights_cuda(torch::Tensor &origin_weight,
                                      bool is_int4)
{
    // guarantee the weight is cpu tensor
    CHECK_CPU(origin_weight);

    torch::Tensor preprocessed_quantized_weight = torch::empty_like(origin_weight);
    int8_t *preprocessed_quantized_weight_ptr = reinterpret_cast<int8_t *>(preprocessed_quantized_weight.data_ptr());
    const int8_t *row_major_quantized_weight_ptr = reinterpret_cast<const int8_t *>(origin_weight.data_ptr());
    size_t rows = origin_weight.size(-2);
    size_t cols = origin_weight.size(-1);
    int arch = ft::getSMVersion();
    ft::preprocess_weights(preprocessed_quantized_weight_ptr,
                                          row_major_quantized_weight_ptr,
                                          rows,
                                          cols,
                                          is_int4,
                                          arch);
    return preprocessed_quantized_weight;
}

void fpA_intB_gemm_forward_cuda(torch::Tensor &input,
                                torch::Tensor &weight,
                                torch::Tensor &scale,
                                torch::Tensor &output,
                                int m, int n, int k)
{
    c10::cuda::CUDAGuard device_guard(input.device());
    const fastertransformer::half *input_ptr = reinterpret_cast<fastertransformer::half *>(input.data_ptr());
    const uint8_t *weight_ptr = reinterpret_cast<const uint8_t *>(weight.data_ptr());
    const fastertransformer::half *scale_ptr = reinterpret_cast<fastertransformer::half *>(scale.data_ptr());
    fastertransformer::half *output_ptr = reinterpret_cast<fastertransformer::half *>(output.data_ptr());
    // const int max_size = std::max(n, k);
    // size_t workspace_size = getWorkspaceSize(m, max_size, max_size);
    // void *ptr = nullptr;
    // char *workspace_ptr = workspace_size > 0 ? (char *)hipMalloc((void **)&ptr, workspace_size) : nullptr;

    fastertransformer::gemm_fp16_int_bias_act(
        input_ptr,
        weight_ptr,
        scale_ptr,
        nullptr,
        output_ptr,
        std::nullopt,
        m, n, k,
        0,
        nullptr,
        0,
        0);
}